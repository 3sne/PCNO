#include "hip/hip_runtime.h"

#include "utils.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void ReduceSum(double* A, double* B, int dim) {
    
    
    if(dim == 1)
    {
        int row = blockIdx.x;
        B[row] = 0;
        if (threadIdx.x == 0) {
            for(int  i =0;i<blockDim.x;i++) {
                
                B[row]+= A[row*blockDim.x + i]; 
            }
           
        }
    }

}

__global__ void square(double* A, double *B) {
    
    B[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x]*A[blockIdx.x * blockDim.x + threadIdx.x]; 
}

__global__ void transpose(double* A, double* B)
{
   B[threadIdx.x*gridDim.x + blockIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x];
}



__global__ void dot(double* A, double* B, double* C, int col_size) {
    
    for(int i = 0; i < col_size;i++) {
        C[blockIdx.x*blockDim.x + threadIdx.x]+=A[blockIdx.x*col_size + i]*B[i*blockDim.x + threadIdx.x];
    }
}

__global__ void Add(double* A, double* B, double* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] + B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void AddAS(double* A, double* C, double scalar) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] + scalar;

}

__global__ void Sub(double* A, double* B, double* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] - B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void zeros(double* A) {

    A[blockIdx.x * blockDim.x + threadIdx.x] = 0.0;
}

__global__ void ones(double* A) {
    A[blockIdx.x * blockDim.x + threadIdx.x] = 1.0;
}

__global__ void Negative(double* A) {
    A[blockIdx.x * blockDim.x + threadIdx.x] = - A[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void MultiplyAS(double* A, double *B, double scalar) {
   B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x] * scalar;
}

__global__ void Exp(double* A, double* B) {
    B[blockIdx.x * blockDim.x + threadIdx.x] = exp(A[blockIdx.x * blockDim.x + threadIdx.x]);
}

__global__ void Log(double* A, double* B) {
    B[blockIdx.x * blockDim.x + threadIdx.x] = log(A[blockIdx.x * blockDim.x + threadIdx.x]);
}

__global__ void MultiplyAA(double* A, double* B, double* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] * B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void DivideAS(double* A, double* B, double scalar) { 

    B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x] / scalar;
}

__global__ void Divide(double* A, double* B, double* C) { 
    C[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x] / B[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void MaxAS(double* A, double* B, double scalar) {
    if(A[blockIdx.x * blockDim.x + threadIdx.x]>scalar) {
        B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x];
    }else {
        B[blockIdx.x * blockDim.x + threadIdx.x] =  scalar;
    }
}

__global__ void DivideSA(double* A, double* B, double scalar) { 
    B[blockIdx.x * blockDim.x + threadIdx.x] =   scalar / A[blockIdx.x * blockDim.x + threadIdx.x] ;
}

__global__ void SetDiagonal(double* A, double scalar) {
    if(blockIdx.x == threadIdx.x) {
        A[blockIdx.x*blockDim.x+threadIdx.x] = scalar;
    }
}

__global__ void BroadcastArrayToMatrix(double* A, double* B) {
    
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[threadIdx.x];
}

__global__ void Range(double* A, int n ) {
    A[blockIdx.x*blockDim.x + threadIdx.x] = blockIdx.x*blockDim.x + threadIdx.x;
}

__global__ void Randn(double* A) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(4, i, 0, &state);
    A[i] = hiprand_uniform(&state);  
}

__global__ void Copy(double* A, double* B){
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[blockIdx.x*blockDim.x + threadIdx.x];
}

__global__ void Stack(double* A, double* B){
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[threadIdx.x];

}

__global__ void IsGreaterThan(double* A, int* B, double scalar){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] > scalar) {
        B[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        B[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }
}

__global__ void IsEqual(int* A, int* B, int* C){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] == B[blockIdx.x*blockDim.x + threadIdx.x])
    {
        C[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        C[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }
}

__global__ void IsNotEqual(int* A, int* B, int* C){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] != B[blockIdx.x*blockDim.x + threadIdx.x])
    {
        C[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        C[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }
}

__global__ void SetWhereLessThan(double* A, double scalar1, double scalar2){
    if(A[blockIdx.x*blockDim.x + threadIdx.x]<scalar1) {
        A[blockIdx.x*blockDim.x + threadIdx.x] = scalar2;
    }
}



void ReduceSumDriver(double A[], double B[], int rowa, int cola, int dimb, int dim) {
    
    double *d_a, *d_b; 
    hipMalloc((void **) &d_a, sizeof(double)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(double)*dimb);
    hipMemcpy(d_a, A, sizeof(double)*rowa*cola, hipMemcpyHostToDevice);

    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    ReduceSum<<<GridDim, BlockDim>>>(d_a, d_b, dim);
    hipMemcpy(B, d_b, sizeof(double)*dimb, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}



void SquareDriver(double A[], double B[], int rowa, int cola) {
    double *d_a, *d_b;
    hipMalloc((void**)&d_a, sizeof(double)*rowa*cola);
    hipMalloc((void**)&d_b, sizeof(double)*rowa*cola);
    hipMemcpy(d_a, A, sizeof(double)*rowa*cola, hipMemcpyHostToDevice);
    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    square<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*rowa*cola, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void TransposeDriver(double A[], double B[], int rowa, int cola) {
    
    double *d_a, *d_b; 
    hipMalloc((void **) &d_a, sizeof(double)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(double)*rowa*cola);
    hipMemcpy(d_a, A, sizeof(double)*rowa*cola, hipMemcpyHostToDevice);

    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    transpose<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*rowa*cola, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void DotDriver(double A[], double B[], double C[], int rowa, int cola, int rowb, int colb) {
    double *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(double)*rowb*colb);
    hipMalloc((void **) &d_c, sizeof(double)*rowa*colb);
    hipMemcpy(d_a, A, sizeof(double)*rowa*cola, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(double)*rowb*colb, hipMemcpyHostToDevice);
    dim3 BlockDim(colb);
    dim3 GridDim(rowa);
    dot<<<GridDim, BlockDim>>>(d_a, d_b, d_c, cola);
    hipMemcpy(C, d_c, sizeof(double)*rowa*colb, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void AddDriver(double A[], double B[], double C[], int row, int col) {

    double *d_a, *d_b, *d_c; 

    hipMalloc((void **) &d_a, sizeof(double)*row*col);
    hipMalloc((void **) &d_b, sizeof(double)*row*col);
    hipMalloc((void **) &d_c, sizeof(double)*row*col);

    hipMemcpy(d_a, A, sizeof(double)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(double)*row*col, hipMemcpyHostToDevice);

    dim3 BlockDim(col);
    dim3 GridDim(row);
    Add<<<GridDim, BlockDim>>>(d_a, d_b, d_c);

    hipMemcpy(C, d_c, sizeof(double)*row*col, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void SubDriver(double A[], double B[], double C[], int row, int col) {
    
    double *d_a, *d_b, *d_c; 
    

    hipMalloc((void **) &d_a, sizeof(double)*row*col);
    hipMalloc((void **) &d_b, sizeof(double)*row*col);
    hipMalloc((void **) &d_c, sizeof(double)*row*col);
   
    hipMemcpy(d_a, A, sizeof(double)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(double)*row*col, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Sub<<<GridDim, BlockDim>>>(d_a, d_b, d_c);

    hipMemcpy(C, d_c, sizeof(double)*row*col, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void ZerosDriver(double A[], int row, int col) {
    double *d_a;
    hipMalloc((void**)&d_a, sizeof(double)*row*col);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    zeros<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(double)*row*col, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipDeviceSynchronize();
}


void OnesDriver(double A[], int row, int col) {
    double *d_a;
    hipMalloc((void**)&d_a, sizeof(double)*row*col);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    ones<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(double)*row*col, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipDeviceSynchronize();
}


void GetReducedRow(double A[], double B[], int row, int col, int rowtoget, int coltoremove) {
    int b_index = 0;
    for(int i = 0; i<col;i++) {
        if(i==coltoremove) {
            continue;
        }
        B[b_index++]=A[rowtoget*col + i];
    }
}





void NegativeDriver(double A[], int size) {
    double *d_a;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    Negative<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipDeviceSynchronize();
}


void ExpDriver(double A[], double B[], int size) {
    double *d_a, *d_b;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    hipMalloc((void**)&d_b, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    Exp<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();

}


void LogDriver(double A[], double B[], int size) {
    double *d_a, *d_b;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    hipMalloc((void**)&d_b, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    Log<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void ReduceSumDriver(double A[], double* B, int size) {
    for(int i = 0;i<size;i++){
        (*B)+=A[i];
    }  
}



void MultiplyDriver(double A[], double B[], double C[], int size) {
    double *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_b, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    MultiplyAA<<<GridDim, BlockDim>>>(d_a, d_b, d_c);
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void MultiplyDriver(double A[], double B, double C[], int size) {
    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(1);
    dim3 GridDim(size);
    MultiplyAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void DivideDriver(double A[], double B, double C[], int size) {
    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(1);
    dim3 GridDim(size);
    DivideAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}

 
void ReplaceRowExceptCol(double A[], double B[], int row, int col, int rowtoreplace, int colexcept) {
    int b_index = 0;
    for(int i=0;i<col;i++) {
        if(i==colexcept)
            continue;
        A[rowtoreplace*col+i] = B[b_index++];
    }
}


void MaxASDriver(double A[], double B, double C[], int row, int col) {
    int size = row*col;
    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    MaxAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();

}


void DivideDriver( double B, double A[], double C[], int row, int col) {
    int size = row*col;
    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    DivideSA<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}



void SetDiagonalDriver(double A[], double B, int row, int col) {
    int size = row*col;
    double *d_a; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    SetDiagonal<<<GridDim, BlockDim>>>(d_a, B );
    hipMemcpy(A, d_a, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipDeviceSynchronize();
}


void BroadcastArrayToMatrixDriver(double A[], double B[], int row, int col) {
    int size = row*col;
    double *d_a, *d_b; 

    hipMalloc((void **) &d_a, sizeof(double)*col);
    hipMalloc((void**) &d_b, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*col, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    BroadcastArrayToMatrix<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void RangeDriver(double A[], int size, int n) {
    
    double* d_a;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    Range<<<GridDim, BlockDim>>>(d_a, n);
    hipMemcpy(A, d_a, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipFree(d_a);
}


void RandnDriver(double A[], int row, int col) {
    int size = row*col;
    double *d_a;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Randn<<<GridDim, BlockDim>>>(d_a);
    hipFree(d_a);
    hipMemcpy(A, d_a, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
}

void MultiplyDriver(double A[], double B, double C[], int row, int col) {
    int size = row*col;
    
    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    MultiplyAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void CopyDriver(double A[], double B[], int size){
    double *d_a,*d_b;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    hipMalloc((void**)&d_b, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    Copy<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void GetRow(double A[], double B[], int row, int col, int rownum) {
    for(int  i = 0 ;i<col;i++) {
        B[i] = A[rownum*col + i];
    }
}


void SetRow(double A[], double B[], int row, int col, int rownum) {
    for(int  i = 0 ;i<col;i++) {
         A[rownum*col + i] = B[i];
    }
}



void GetCol(double A[], double B[], int row, int col, int colnum) {
    for(int  i = 0 ;i<row;i++) {
        B[i] = A[i*col + colnum];
    }
}


void StackDriver(double A[], double B[], int row, int col){
    int size = row*col;    
    double *d_a,*d_b;
    hipMalloc((void**)&d_a, sizeof(double)*size);
    hipMalloc((void**)&d_b, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Stack<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipDeviceSynchronize();
}


void IsGreaterThanDriver(double A[], double B, int* C, int size){

    
    double *d_a;
    int *d_c;
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(1);
    dim3 GridDim(size);
    IsGreaterThan<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void IsEqualDriver(int A[], int B[], int C[], int size){
    int *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(int)*size);
    hipMalloc((void **) &d_b, sizeof(int)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(int)*size, hipMemcpyHostToDevice);

    dim3 BlockDim(1);
    dim3 GridDim(size);
    IsEqual<<<GridDim, BlockDim>>>(d_a, d_b, d_c );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void IsNotEqualDriver(int A[], int B[], int C[], int size){
    int *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(int)*size);
    hipMalloc((void **) &d_b, sizeof(int)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(int)*size, hipMemcpyHostToDevice);

    dim3 BlockDim(1);
    dim3 GridDim(size);
    IsNotEqual<<<GridDim, BlockDim>>>(d_a, d_b, d_c );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c); 
    hipDeviceSynchronize();
}


void AddDriver(double A[], double B, double C[], int size) {

    double *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(1);
    dim3 GridDim(size);
    AddAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void MultiplyDriver(double A[], int B[], double C[], int size){
    double B_fl[size];
    for(int  i = 0; i<size;i++ ){
        B_fl[i] = (double)B[i];
    }
    MultiplyDriver(A, B_fl, C, size);
}


void SetWhereLessThanDriver(double A[], double scalar1, double scalar2, int size){
    double *d_a; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    
    
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(1);
    dim3 GridDim(size);
    SetWhereLessThan<<<GridDim, BlockDim>>>(d_a, scalar1, scalar2 );
    hipMemcpy(A, d_a, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipDeviceSynchronize();
}


void ReduceMeanDriver(double A[], double B[], int row, int col, int dimb, int dim){
    double *d_a, *d_b, *d_c;
   
    hipMalloc((void **) &d_a, sizeof(double)*row*col);
    hipMalloc((void **) &d_b, sizeof(double)*dimb);
    hipMalloc((void **) &d_c, sizeof(double)*dimb);
    hipMemcpy(d_a, A, sizeof(double)*row*col, hipMemcpyHostToDevice);

    dim3 BlockDim(col);
    dim3 GridDim(row);
    ReduceSum<<<GridDim, BlockDim>>>(d_a, d_b, dim);
     
    DivideAS<<<1, col>>>(d_b, d_c, row);
    hipMemcpy(B, d_c, sizeof(double)*row, hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}


void DivideDriver(double A[], double B[], double C[], int size){
    double *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(double)*size);
    hipMalloc((void **) &d_b, sizeof(double)*size);
    hipMalloc((void **) &d_c, sizeof(double)*size);
    hipMemcpy(d_a, A, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(double)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(1);
    dim3 GridDim(size);
    Divide<<<GridDim, BlockDim>>>(d_a, d_b, d_c);
    hipMemcpy(C, d_c, sizeof(double)*size, hipMemcpyDeviceToHost); 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipDeviceSynchronize();
}