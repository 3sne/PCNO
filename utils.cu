#include "hip/hip_runtime.h"
extern "C" {
#include "utils.h"
}


__global__ void ReduceSum(float* A, float* B, int dim) {
    
    
    if(dim == 1)
    {
        int row = blockIdx.x;
        B[row] = 0;
        if (threadIdx.x == 0) {
            for(int  i =0;i<blockDim.x;i++) {
                
                B[row]+= A[row*blockDim.x + i]; 
            }
           
        }
    }else if(dim==0) {
        int col = threadIdx.x;
        if(blockIdx.x == 0) {
            for(int i=0;i<GridDim.x;i++) {
                B[col]+= A[i * blockIdx.x + col];
            }
        }
    }else {
        __global__ float stash_sum[gridDim.x];
        int row = blockIdx.x;
        stash_sum[row] = 0;
        if (threadIdx.x == 0) {
            for(int  i =0;i<blockDim.x;i++) {
                
                stash_sum[row]+= A[row*blockDim.x + i]; 
            }
            *B = 0;
            for(int i = 0;i<gridDim.x;i++) {
                (*B)+= stash_sum[i];
            }
           
        }
        
    }

}

__global__ void square(float* A, float *B) {
    
    B[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x]*A[blockIdx.x * blockDim.x + threadIdx.x]; 
}

__global__ void transpose(float* A, float* B)
{
   B[threadIdx.x*gridDim.x + blockIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x];
}



__global__ void dot(float* A, float* B, float* C, int col_size) {
    
    for(int i = 0; i < col_a;i++) {
        C[row*blockDim.x + threadIdx.x]+=A[row*col_size + i]*B[i*blockDim.x + threadIdx.x];
    }
}

__global__ void Add(float* A, float* B, float* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] + B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void AddAS(float* A, float* C, float scalar) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] + scalar;

}

__global__ void Sub(float* A, float* B, float* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] - B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void zeros(float* A) {

    A[blockIdx.x * blockDim.x + threadIdx.x] = 0.0;
}

__global__ void ones(float* A) {
    A[blockIdx.x * blockDim.x + threadIdx.x] = 1.0;
}

__global__ void Negative(float* A) {
    A[blockIdx.x * blockDim.x + threadIdx.x] = - A[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void MultiplyAS(float* A, float *B, float scalar) {
   B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x] * scalar;
}

__global__ void Exp(float* A, float* B) {
    B[blockIdx.x * blockDim.x + threadIdx.x] = expf(A[blockIdx.x * blockDim.x + threadIdx.x]);
}

__global__ void MultiplyAA(float* A, float* B, float* C) {
   

    C[blockIdx.x * blockDim.x + threadIdx.x] = A[blockIdx.x * blockDim.x + threadIdx.x] * B[blockIdx.x * blockDim.x + threadIdx.x];

}

__global__ void DivideAS(float* A, float* B, float scalar) { 
    B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x] / scalar;
}

__global__ void MaxAS(float* A, float* B, float scalar) {
    if(A[blockIdx.x * blockDim.x + threadIdx.x]>scalar) {
        B[blockIdx.x * blockDim.x + threadIdx.x] =  A[blockIdx.x * blockDim.x + threadIdx.x];
    }else {
        B[blockIdx.x * blockDim.x + threadIdx.x] =  scalar;
    }
}

__global__ void DivideSA(float* A, float* B, float scalar) { 
    B[blockIdx.x * blockDim.x + threadIdx.x] =   scalar / A[blockIdx.x * blockDim.x + threadIdx.x] ;
}

__global__ void SetDiagonal(float* A, float scalar) {
    if(blockIdx.x == threadIdx.x) {
        A[blockIdx.x*blockDim.x+threadIdx.x] = scalar;
    }
}

__global__ void BroadcastArrayToMatrix(float* A, float* B) {
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[blockIdx.x];
}

__global__ void Range(float* A, int n ) {
    A[blockIdx.x*blockDim.x + threadIdx.x] = blockIdx.x*blockDim.x + threadIdx.x;
}

__global__ void Randn(float* A) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init(seed, i, 0, &state);
    A[i] = hiprand_uniform(&state);  
}

__global__ void Copy(float* A, float* B){
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[blockIdx.x*blockDim.x + threadIdx.x];
}

__global__ void Stack(float* A, float *B){
    B[blockIdx.x*blockDim.x + threadIdx.x] = A[threadIdx];

}

__global__ void IsGreaterThan(float* A, int* B, float scalar){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] > scalar) {
        B[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        B[blockIdx.x*blockDim.x + threadIdx.x] = 0
    }
}

__global__ void IsEqual(int* A, int* B, int* C){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] == B[blockIdx.x*blockDim.x + threadIdx.x])
    {
        C[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        C[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }
}

__global__ void IsNotEqual(int* A, int* B, int* C){
    if(A[blockIdx.x*blockDim.x + threadIdx.x] != B[blockIdx.x*blockDim.x + threadIdx.x])
    {
        C[blockIdx.x*blockDim.x + threadIdx.x] = 1;
    }else{
        C[blockIdx.x*blockDim.x + threadIdx.x] = 0;
    }
}

__global__ SetWhereLessThan(float A[], float scalar1, float scalar2){
    if(A[blockIdx.x*blockDim.x + threadIdx.x]<scalar1) {
        B[blockIdx.x*blockDim.x + threadIdx.x] = scalar2;
    }
}


extern "C"
void ReduceSumDriver(float A[], float B[], int rowa, int cola, int dimb, int dim) {
    
    float *d_a, *d_b; 
    hipMalloc((void **) &d_a, sizeof(float)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(float)*dimb);
    hipMemcpy(d_a, A, sizeof(float)*rowa*cola, hipMemcpyHostToDevice);

    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    ReduceSum<<<GridDim, BlockDim>>>(d_a, d_b, dim);
    hipMemcpy(B, d_b, sizeof(float)*dimb, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}


extern "C"
void SquareDriver(float A[], float B[], int rowa, int cola) {
    float *d_a, *d_b;
    hipMalloc((void**)&d_a, sizeof(float)*rowa*cola);
    hipMalloc((void**)&d_b, sizeof(float)*rowa*cola);
    hipMemcpy(d_a, A, sizeof(float)*rowa*cola, hipMemcpyHostToDevice);
    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    square<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*rowa*cola, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void TransposeDriver(float A[], float B[], int rowa, int cola) {
    
    float *d_a, *d_b; 
    hipMalloc((void **) &d_a, sizeof(float)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(float)*rowa*cola);
    hipMemcpy(d_a, A, sizeof(float)*rowa*cola, hipMemcpyHostToDevice);

    dim3 BlockDim(cola);
    dim3 GridDim(rowa);
    transpose<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*rowa*cola, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void DotDriver(float A[], float B[], float C[], int rowa, int cola, int rowb, int colb) {
    float *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*rowa*cola);
    hipMalloc((void **) &d_b, sizeof(float)*rowb*colb);
    hipMalloc((void **) &d_c, sizeof(float)*rowa*colb);
    hipMemcpy(d_a, A, sizeof(float)*rowa*cola, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(float)*rowb*colb, hipMemcpyHostToDevice);
    dim3 BlockDim(colb);
    dim3 GridDim(rowa);
    dot<<<GridDim, BlockDim>>>(d_a, d_b, cola);
    hipMemcpy(C, d_c, sizeof(float)*rowa*colb, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void AddDriver(float A[], float B[], float C[], int row, int col) {
    float *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*row*col);
    hipMalloc((void **) &d_b, sizeof(float)*row*col);
    hipMalloc((void **) &d_c, sizeof(float)*row*col);
    hipMemcpy(d_a, A, sizeof(float)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(float)*row*col, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Add<<<GridDim, BlockDim>>>(d_a, d_b, d_c);
    hipMemcpy(C, d_c, sizeof(float)*row*col, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void SubDriver(float A[], float B[], float C[], int row, int col) {
    float *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*row*col);
    hipMalloc((void **) &d_b, sizeof(float)*row*col);
    hipMalloc((void **) &d_c, sizeof(float)*row*col);
    hipMemcpy(d_a, A, sizeof(float)*row*col, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(float)*row*col, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Add<<<GridDim, BlockDim>>>(d_a, d_b, d_c);
    hipMemcpy(C, d_c, sizeof(float)*row*col, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void ZerosDriver(float A[], int row, int col) {
    float *d_a;
    hipMalloc((void**)&d_a, sizeof(float)*rowa*cola);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    zeros<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(float)*row*col, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void OnesDriver(float A[], int row, int col) {
    float *d_a;
    hipMalloc((void**)&d_a, sizeof(float)*row*col);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    ones<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(float)*row*col, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void GetReducedRow(float A[], float B[], int row, int col, int rowtoget, int coltoremove) {
    int b_index = 0;
    for(int i = 0; i<col;i++) {
        if(i==coltoremove) {
            continue;
        }
        B[b_index++]=A[rowtoget*col + i];
    }
}



extern "C"

void NegativeDriver(float A[], int size) {
    float *d_a;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    Negative<<<GridDim, BlockDim>>>(d_a);
    hipMemcpy(A, d_a, sizeof(float)*size, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void ExpDriver(float A[], float B[], int size) {
    float *d_a, *d_b;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    hipMalloc((void**)&d_b, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    Exp<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*size, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();
}

extern "C"
void ReduceSumDriver(float A[], float* B, int size) {
    float *d_a;
    hipMalloc((void**)&d_a, sizeof(float)*size);
   
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    ReduceSum<<<GridDim, BlockDim>>>(d_a, B, -1);
    hipMemcpy(B, d_b, sizeof(float)*size, hipMemcpyDeviceToHost); 
    
    hipDeviceSynchronize();   
}


extern "C"
void MultiplyDriver(float A[], float B[], float C[], int size) {
    float *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_b, sizeof(float)*size);
    hipMalloc((void **) &d_c, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    MultiplyAA<<<GridDim, BlockDim>>>(d_a, d_b, d_c);
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void MultiplyDriver(float A[], float B, float C[], int size) {
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_c, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(size);
    dim3 GridDim(1);
    MultiplyAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void DivideDriver(float A[], float B, float C[], int size) {
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_b, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(size);
    dim3 GridDim(1);
    DivideAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C" 
void ReplaceRowExceptCol(float A[], float B[] int row, int col, int rowtoreplace, int colexcept) {
    int b_index = 0;
    for(int i=0;i<col;i++) {
        if(i==colexcept)
            continue;
        A[rowtoreplace*col+i] = B[b_index++];
    }
}

extern "C"
void MaxASDriver(float A[], float B, float C[], int row, int col) {
    int size = row*col;
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_b, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    DivideAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();

}

extern "C"
void DivideDriver( float B, float A[], float C[], int row, int col) {
    int size = row*col;
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_b, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    DivideSA<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}


extern "C"
void SetDiagonalDriver(float A[], float B, int row, int col) {
    int size = row*col;
    float *d_a; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    SetDiagonal<<<GridDim, BlockDim>>>(d_a, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void BroadcastArrayToMatrixDriver(float A[], float B[], int row, int col) {
    int size = row*col;
    float *d_a, *d_b; 
    hipMalloc((void **) &d_a, sizeof(float)*row);
    hipMalloc((void**) &d_b, sizeof(float)*size)
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    BroadcastArrayToMatrix<<<col, row>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}

extern "C"
void RangeDriver(float A[], int size, int n) {
    
    float* d_a;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    Range<<<GridDim, BlockDim>>>(A, n);
    hipMemcpy(A, d_a, sizeof(float)*size, hipMemcpyDeviceToHost);
}

extern "C"
void RandnDriver(float A[], int row, int col) {
    int size = row*col;
    float *d_a;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Randn<<<GridDim, BlockDim>>>(A);
    hipMemcpy(A, d_a, sizeof(float)*size, hipMemcpyDeviceToHost); 
}

void MultiplyDriver(float A[], float B, float C, int row, int col) {
    int size = row*col;
    
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_c, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(col);
    dim3 GridDim(row);
    MultiplyAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void CopyDriver(float A[], float B[], int size){
    float *d_a,*d_b;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    hipMalloc((void**)&d_b, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(size);
    dim3 GridDim(1);
    Copy<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void GetRow(float A[], float B[], int row, int col, int rownum) {
    for(int  i = 0 ;i<col;i++) {
        B[i] = A[rownum*col + i];
    }
}

extern "C"
void SetRow(float A[], float B[], int row, int col, int rownum) {
    for(int  i = 0 ;i<col;i++) {
         A[rownum*col + i] = B[i];
    }
}


extern "C"
void GetCol(float A[], float B[], int row, int col, int colnum) {
    for(int  i = 0 ;i<col;i++) {
        B[i] = A[i*col + colnum];
    }
}

extern "C"
void StackDriver(float A[], float B[], int row, int col){
    int size = row*col;    
    float *d_a,*d_b;
    hipMalloc((void**)&d_a, sizeof(float)*size);
    hipMalloc((void**)&d_b, sizeof(float)*size);
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    dim3 BlockDim(col);
    dim3 GridDim(row);
    Stack<<<GridDim, BlockDim>>>(d_a, d_b);
    hipMemcpy(B, d_b, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void IsGreaterThanDriver(float A[], float B, int* C, int size){

    
    float *d_a;
    int *d_c 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(size);
    dim3 GridDim(1);
    IsGreaterThan<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void IsEqualDriver(int A[], int B[], int C[], int size){
    int *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(int)*size);
    hipMalloc((void **) &d_b, sizeof(int)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(int)*size, hipMemcpyHostToDevice);

    dim3 BlockDim(size);
    dim3 GridDim(1);
    IsEqual<<<GridDim, BlockDim>>>(d_a, d_b, d_c );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void IsNotEqualDriver(int A[], int B[], int C[], int size){
    int *d_a, *d_b, *d_c; 
    hipMalloc((void **) &d_a, sizeof(int)*size);
    hipMalloc((void **) &d_b, sizeof(int)*size);
    hipMalloc((void **) &d_c, sizeof(int)*size);
    
    hipMemcpy(d_a, A, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sizeof(int)*size, hipMemcpyHostToDevice);

    dim3 BlockDim(size);
    dim3 GridDim(1);
    IsNotEqual<<<GridDim, BlockDim>>>(d_a, d_b, d_c );
    hipMemcpy(C, d_c, sizeof(int)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void AddDriver(float A[], float B, float C[], int size) {
    float *d_a, *d_c; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    hipMalloc((void **) &d_c, sizeof(float)*size);
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(size);
    dim3 GridDim(1);
    AddAS<<<GridDim, BlockDim>>>(d_a, d_c, B );
    hipMemcpy(C, d_c, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void MultiplyDriver(float A[], int B[], float C[], int size){
    float B_fl[size];
    for(int  i = 0; i<size;i++ ){
        B_fl[i] = (float)B[i];
    }
    MultiplyDriver(A, B_fl, C, size);
}

extern "C"
void SetWhereLessThanDriver(float A[], float scalar1, float scalar2, int size){
    float *d_a; 
    hipMalloc((void **) &d_a, sizeof(float)*size);
    
    
    hipMemcpy(d_a, A, sizeof(float)*size, hipMemcpyHostToDevice);
    
    dim3 BlockDim(size);
    dim3 GridDim(1);
    SetWhereLessThan<<<GridDim, BlockDim>>>(d_a, scalar1, scalar2 );
    hipMemcpy(A, d_a, sizeof(float)*size, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
}

extern "C"
void ReduceMeanDriver(float A[], float B[], int row, int col, int dimb, int dim){
    float *d_a, *d_b, *d_c;
    float C[col]; 
    hipMalloc((void **) &d_a, sizeof(float)*row*col);
    hipMalloc((void **) &d_b, sizeof(float)*dimb);
    hipMalloc((void **) &d_c, sizeof(float)*dimb);
    hipMemcpy(d_a, A, sizeof(float)*rowa*cola, hipMemcpyHostToDevice);

    dim3 BlockDim(col);
    dim3 GridDim(row);
    ReduceSum<<<GridDim, BlockDim>>>(d_a, d_b, dim);
     
    DivideAS<<<1, col>>>(d_b, d_c, row);
    hipMemcpy(B, d_c, sizeof(float)*size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}
